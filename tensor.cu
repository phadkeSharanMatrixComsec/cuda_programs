#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void assign(int **tensor, int N, int M, int value)
{

    // int i = blockIdx.x * blockDim.x + threadIdx.x;
    // int j = blockIdx.y * blockDim.y + threadIdx.y;

    // if(i < N && j < N)
    // {
    //     tensor[i][j] = 88;
    //     printf("Assigned\n");
    // }
    for(int i=0;i<N;i++)
    {
        for(int j=0;j<M;j++)
        {
            tensor[i][j] = value;
        }
    }

}

__global__ void printTensor(int **tensor, int N, int M)
{

    // int i = blockIdx.x * blockDim.x + threadIdx.x;
    // int j = blockIdx.y * blockDim.y + threadIdx.y;

    // if(i < N && j < N)
    // {
    //     printf("%d ", tensor[i][j]);
    //     printf("print\n");
    // }
    
    for(int i=0;i<N;i++)
    {
        for(int j=0;j<M;j++)
        {
            printf("%d ", tensor[i][j]);
        }

        printf("\n");
    }

}

// __host__ __device__ void tensorAllocate(int **tensor, int N, int M)
// {
//     cudaMallocManaged((int ***)&tensor, N * sizeof(int));
//     for(int i=0;i<N;i++)
//     {
//         cudaMallocManaged((int **)&tensor[i], M * sizeof(int));
//     }
// }

int main()
{
    int **tensor;
    int N, M, value;

    while(1)
    {
        scanf("%d", &N);
        scanf("%d", &M);
        scanf("%d", &value);

        hipDeviceSetLimit(hipLimitMallocHeapSize, 1024 * 1024 * 1024);

        hipError_t e1, e2, e3;

        e1 = hipMallocManaged((int ***)&tensor, N * sizeof(int));

        if(e1 != hipSuccess)
        {
            printf("e1! \n");
        }

        for(int i=0;i<N;i++)
        {
            e2 = hipMallocManaged((int **)&tensor[i], M * sizeof(int));
        }

        if(e2 != hipSuccess)
        {
            printf("e2! \n");
        }

        

        assign<<<1, 1>>>(tensor, N, M, value);
        hipDeviceSynchronize();

        printTensor<<<1, 1>>>(tensor, N, M);
        hipDeviceSynchronize();

        e3 = hipGetLastError();

        if(e3 != hipSuccess)
        {
            printf("%s", hipGetErrorString(e3));
        }
    }
}