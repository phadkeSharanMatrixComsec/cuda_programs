
#include <hip/hip_runtime.h>
#include <stdlib.h>


#define N 10000000


__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out; 
    float *gpuA, *gpuB, *gpuOut;

    a = (float*)malloc(sizeof(float)*N);
    b = (float*)malloc(sizeof(float)*N);
    out = (float*)malloc(sizeof(float)*N);

    hipMalloc(&gpuA, sizeof(float)*N);
    hipMalloc(&gpuB, sizeof(float)*N); 
    hipMalloc(&gpuOut, sizeof(float)*N); 

    for(int i=0;i<N;i++)
    {
        a[i] = 3.2f;
        b[i] = 2.3f;
    }

    hipMemcpy(gpuA, a, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(gpuB, b, sizeof(float)*N, hipMemcpyHostToDevice);

    vector_add<<<1, 1>>>(gpuOut, gpuA, gpuB, N);

    hipFree(gpuA);
    hipFree(gpuB);
    hipFree(gpuOut);

    free(a);
    free(b);
    free(out);
}