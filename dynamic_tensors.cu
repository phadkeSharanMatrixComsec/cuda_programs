#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void printTensor(float *tensor, int N)
{
    printf("working!!\n");
    for(int i=0;i<N;i++)
    {
        for(int j=0;j<N;j++)
        {
            printf("%f ", *(tensor + i));
        }

        printf("\n");
    }

}

__global__ void printArray(float *ptr, int N)
{
    for(int i=0;i<N;i++)
    {
        printf("%d ", *(ptr+i));
    }
    printf("\n");
}

int main()
{
    int N=10;
    float arr[10][10];
    float *tensor;

    for(int i=0;i<N;i++)
    {
        for(int j=0;j<N;j++)
        {
            arr[i][j] = 10.0;
        }
    }

    // for(int i=0;i<N;i++)
    // {
    //     for(int j=0;j<N;j++)
    //     {
    //         printf("%f ", arr[i][j]);
    //     }
    // }

    hipError_t err2 = hipMalloc(&tensor, sizeof(float) * N * N);
    hipError_t err = hipMemcpy(tensor, arr, sizeof(float) * N * N, hipMemcpyHostToDevice);

    if(err != hipSuccess || err2 != hipSuccess)
    {
        printf("loccha!");
    }

    // printTensor<<<1, 1>>>(tensor, N);
    printArray<<<1, 1>>>(tensor, N*N);

}